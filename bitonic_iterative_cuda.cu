#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <time.h>
#include <omp.h>
#include <stdlib.h>
#include <assert.h>

#include "util.h"


template <typename T> 
__device__ void compare_and_swap(T * a, T * b, int inc) {
  if ((*a > *b && inc) || (*a < *b && !inc)) {
    T tmp = *a;
    *a = *b;
    *b = tmp;
  }
}

template <typename T> 
__global__ void bitonic_merge(T * arr, int split_size, int bm_size) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int cmp_idx = idx ^ split_size;
  if (idx < cmp_idx) {
    compare_and_swap<T>(&arr[idx], &arr[cmp_idx], (idx & bm_size) == 0);
  }

}

int is_power_of_2(int x) {
  return x > 0 && (((x-1) & x) == 0);
}

template <typename T> 
void cuda_main(int size) {
  int num_bytes = size * sizeof(T);
  
  // change
  T * arr = gen_data_real<T>(size, 0, std::numeric_limits<T>::max());
  
  T * d_arr;

  hipMalloc((void **)&d_arr, num_bytes);

  int num_blocks = size / 512;
  int num_threads = 512;

  double tstart, tend;
  tstart = omp_get_wtime();
  hipMemcpy(d_arr, arr, num_bytes, hipMemcpyHostToDevice);
  for (int bm_size = 2; bm_size <= size; bm_size *= 2) {
    for (int split_size = bm_size / 2; split_size >= 1; split_size /= 2) {
      bitonic_merge<<<num_blocks, num_threads>>>(d_arr, split_size, bm_size);
    }
  }
  hipMemcpy(arr, d_arr, num_bytes, hipMemcpyDeviceToHost);
  tend = omp_get_wtime();

  printf("Time: %f \n", tend - tstart);

  for (int i = 0; i < size - 1; i++) {
    if (arr[i] > arr[i+1]) {
      std::cout << "wrong" << std::endl;
      return;
    }
  }
  std::cout << "correct" << std::endl;

  delete[] arr;
}

int main(int argc, char * argv[]) {
  // srand(time(NULL));
  srand(0);

  if (argc < 2) {
    std::cout << "bitonic <# elements>" << std::endl;
    return 0;
  }
  int size = atoi(argv[1]);
  if (!is_power_of_2(size)) {
    std::cout << "Array size must be power of 2" << std::endl;
    return -1;
  }
  if (size <= 512) {
    std::cout << "Array size must larger than 512" << std::endl;
    return -1;
  }

  cuda_main<double>(size);

  return 0;

}